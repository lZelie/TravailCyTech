#include "hip/hip_runtime.h"
﻿#include <array>
#include <cassert>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <>
#include <iomanip>
#include <iostream>
#include <random>
#include <sstream>
#include <GL/freeglut_std.h>

#define CHECK_CUDA_ERROR(err) check_cuda_error_d(err, __FILE__, __LINE__)

enum class render_mode: std::uint8_t
{
	cpu,
	gpu,
};

constexpr std::size_t screen_x = 1024;
constexpr std::size_t screen_y = 768;
constexpr std::size_t fps_update = 500;
constexpr auto title = "Bugs";

std::array<float4, screen_x * screen_y> pixels{};
std::size_t frame = 0;
std::size_t time_base = 0;
float scale = 3e-3f;
constexpr int range = 1;
constexpr int survive_low = 3;
constexpr int survive_high = 4;
constexpr int birth_low = 3;
constexpr int birth_high = 3;
__device__ float ambient_light = 0.2f;

auto render_mode = render_mode::cpu;

inline void check_cuda_error_d(const hipError_t err, const std::string& file, const int line)
{
	if (err != hipError_t::hipSuccess)
	{
		std::cerr << file << "(" << line << "): CUDA Runtime API error " << err << ": " << hipGetErrorString(err) <<
			". \n";
		std::cin.get();

		std::quick_exit(err);
	}
}


void init_gl(int argc, char* argv[])
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowPosition(0, 0);
	glutInitWindowSize(screen_x, screen_y);
	glutCreateWindow(title);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glDisable(GL_DEPTH_TEST);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, screen_x, screen_y, 0, 0, 1);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.375, 0.375, 0); // Displacement trick for exact pixelization
}

__global__ void calculate_pixels_gpu_1(const float4* pixels_in_d, float4* pixels_out_d)
{
	const std::size_t index_x = threadIdx.x + blockIdx.x * blockDim.x;
	const std::size_t index_y = threadIdx.y + blockIdx.y * blockDim.y;
	const std::size_t index_pixel = index_y * screen_x + index_x;

	const int min_x = static_cast<int>(index_x) - range;
	const int min_y = static_cast<int>(index_y) - range;

	int livings = 0;
	if (pixels_in_d[index_pixel].y == 1.0f)
	{
		livings -= 1;
	}

	for (int square_x = min_x; square_x < min_x + (2 * range + 1); ++square_x)
	{
		for (int square_y = min_y; square_y < min_y + (2 * range + 1); ++square_y)
		{
			const int corrected_x = square_x >= 0
				                        ? (square_x < static_cast<int>(screen_x)
					                           ? square_x
					                           : square_x -
					                           static_cast<int>(screen_x))
				                        : static_cast<int>(screen_x) + square_x;
			const int corrected_y = square_y >= 0
				                        ? (square_y < static_cast<int>(screen_y)
					                           ? square_y
					                           : square_y -
					                           static_cast<int>(screen_y))
				                        : static_cast<int>(screen_y) + square_y;

			const std::size_t index_pixel_square = corrected_y * screen_x + corrected_x;

			if (pixels_in_d[index_pixel_square].y == 1.0f)
			{
				livings += 1;
			}
		}
	}

	if (pixels_in_d[index_pixel].y == 0.0f && livings >= birth_low && livings <= birth_high)
	{
		pixels_out_d[index_pixel] = make_float4(0.0f, 1.0f, 0.0f, 0.0f);
	}
	else if (pixels_in_d[index_pixel].y == 1.0f && (livings < survive_low || livings > survive_high))
	{
		pixels_out_d[index_pixel] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	}
}


void calculate_pixels_gpu()
{
	/*static int grid = 1;

	if (grid == 1)
	{*/
	float4 *grid1, *grid2;

	hipMalloc(&grid1, screen_x * screen_y * sizeof(float4));
	hipMalloc(&grid2, screen_x * screen_y * sizeof(float4));
	hipMemcpy(grid1, pixels.data(), screen_x * screen_y * sizeof(float4), hipMemcpyHostToDevice);
	// Using grid1 to grid2
	calculate_pixels_gpu_1 <<< dim3(screen_x / 16, screen_y / 16), dim3(16, 16) >>>(grid1, grid2);
	hipMemcpy(pixels.data(), grid2, screen_x * screen_y * sizeof(float4), hipMemcpyDeviceToHost);

	/*grid = 2;*/
	/*}
	else
	{
		// Using grid2 to grid1
		calculate_pixels_gpu_1 <<< dim3(screen_x / 16, screen_y / 16), dim3(16, 16) >>>(grid2, grid1);
		hipMemcpy(pixels.data(), grid1, screen_x * screen_y * sizeof(float4), hipMemcpyDeviceToHost);

		grid = 1;
	}*/
}


void calculate_pixels_cpu()
{
	for (std::size_t index_x = 0; index_x < screen_x; ++index_x)
	{
		for (std::size_t index_y = 0; index_y < screen_y; ++index_y)
		{
			const std::size_t index_pixel = index_y * screen_x + index_x;

			const int min_x = static_cast<int>(index_x) - range;
			const int min_y = static_cast<int>(index_y) - range;

			int livings = 0;
			if (pixels[index_pixel].y == 1.0f)
			{
				livings -= 1;
			}

			for (int square_x = min_x; square_x < min_x + (2 * range + 1); ++square_x)
			{
				for (int square_y = min_y; square_y < min_y + (2 * range + 1); ++square_y)
				{
					const int corrected_x = square_x >= 0
						                        ? (square_x < static_cast<int>(screen_x)
							                           ? square_x
							                           : square_x -
							                           static_cast<int>(screen_x))
						                        : static_cast<int>(screen_x) + square_x;
					const int corrected_y = square_y >= 0
						                        ? (square_y < static_cast<int>(screen_y)
							                           ? square_y
							                           : square_y -
							                           static_cast<int>(screen_y))
						                        : static_cast<int>(screen_y) + square_y;

					const std::size_t index_pixel_square = corrected_y * screen_x + corrected_x;

					if (pixels[index_pixel_square].y == 1.0f)
					{
						livings += 1;
					}
				}
			}

			if (pixels[index_pixel].y == 0.0f && livings >= birth_low && livings <= birth_high)
			{
				pixels[index_pixel] = make_float4(0.0f, 1.0f, 0.0f, 0.0f);
			}
			else if (pixels[index_pixel].y == 1.0f && (livings < survive_low || livings > survive_high))
			{
				pixels[index_pixel] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
			}
		}
	}
}

void calculate()
{
	frame++;

	const int time_current = glutGet(GLUT_ELAPSED_TIME);

	if (time_current - time_base > fps_update)
	{
		std::stringstream ss;
		ss << title << ": " << std::fixed << std::setprecision(3) << static_cast<float>(frame) * 1e3f / static_cast<
			float>(time_current - time_base) << " FPS";
		glutSetWindowTitle(ss.str().c_str());
		time_base = time_current;
		frame = 0;
	}

	if (render_mode == render_mode::cpu)
	{
		calculate_pixels_cpu();
	}
	else
	{
		calculate_pixels_gpu();
	}
}

void render()
{
	calculate();

	glDrawPixels(screen_x, screen_y, GL_RGBA, GL_FLOAT, pixels.data());

	glutSwapBuffers();
}

void idle()
{
	glutPostRedisplay();
}

float3 random_float3(const float min, const float max)
{
	static std::random_device rng;
	std::uniform_real_distribution<float> dist(min, max);
	return {dist(rng), dist(rng), dist(rng)};
}

float random_float(const float min, const float max)
{
	static std::random_device rng;
	std::uniform_real_distribution<float> dist(min, max);
	return dist(rng);
}


void process_normal_keys(const unsigned char key, int, int)
{
	switch (key) // NOLINT(hicpp-multiway-paths-covered)
	{
	case 27:
		std::quick_exit(0);
	case '1':
		render_mode = render_mode::cpu;
		glutPostRedisplay();
		break;
	case '2':
		render_mode = render_mode::gpu;
		glutPostRedisplay();
		break;
	default:
		break;
	}
}

void init()
{
	static std::random_device rng;
	std::uniform_int_distribution<> dist(0, 1);
	for (auto& pixel : pixels)
	{
		pixel.y = dist(rng);
	}
}


int main(const int argc, char* argv[])
{
	init_gl(argc, argv);
	init();

	glutDisplayFunc(render);
	glutIdleFunc(idle);
	glutKeyboardFunc(process_normal_keys);


	glutMainLoop();

	return EXIT_SUCCESS;
}
